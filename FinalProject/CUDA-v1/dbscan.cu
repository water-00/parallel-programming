#include "hip/hip_runtime.h"
#include "dbscan.h"
#include <math.h>
#include <queue>
#include<iostream>
#include "hip/hip_runtime.h"
#include ""
using namespace std;

/*
// calculate eculidean distance of two 2-D points
double euclidean_distance(Point a, Point b)
{
    double x = a.x-b.x;
    double y = a.y-b.y;
    return sqrt(x*x+y*y);
}

// get neighborhood of point p and add it to neighborhood queue

int region_query( vector<Point> &dataset, int p, queue<int> &neighborhood, double eps)
{
	//int count = 0;
    for (int i = 0; i < dataset.size(); i++) {
		//cout << "regin_query" << count++ << endl;
        if(i!=p){
            int dist = euclidean_distance(dataset[p],dataset[i]);
            if ( dist< eps) {
                neighborhood.push(i);
            }
        }
    }
    return (int)neighborhood.size();
}
*/

unsigned int total=0;
__global__ void queryKernel(double *dev_nodeX, double *dev_nodeY, int *dev_result,int* dev_p,double* dev_eps)  
{  
    int i = threadIdx.x;  
	dev_result[i]=1;
	if(i!=*dev_p)
	{
		//int dist=euclidean_distance(dev_nodeX[i],dev_nodeY[i],dev_nodeX[*dev_p],dev_nodeY[*dev_p]);
		double x=dev_nodeX[i]-dev_nodeX[*dev_p];
		double y=dev_nodeY[i]-dev_nodeY[*dev_p];
		int dist=sqrt(x*x+y*y);
		if(dist<*dev_eps)
		{
			dev_result[i]=-9999;
		}
	}
}  

int region_query( vector<Point> &dataset, int p, queue<int> &neighborhood, double eps)
{
	int size=dataset.size();
	
	double* dev_nodeX;
	double* dev_nodeY;
	int* dev_result;
	int* dev_p;
	double* dev_eps;
	
	int *result=(int*)malloc(size*sizeof(int));
	double *nodeX = (double*)malloc(size*sizeof(double));
	double *nodeY = (double*)malloc(size*sizeof(double));
	int i;
	for(i=0;i<size;i++)
	{
		nodeX[i]=dataset[i].x;
		nodeY[i]=dataset[i].y;
	}
	
	hipMalloc((void**)&dev_nodeX,size*sizeof(double));
	hipMalloc((void**)&dev_nodeY,size*sizeof(double));
	hipMalloc((void**)&dev_result,size*sizeof(int));
	hipMalloc((void**)&dev_p,sizeof(int));
	hipMalloc((void**)&dev_eps,sizeof(double));
	hipMemcpy(dev_nodeX,nodeX,size*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_nodeY,nodeY,size*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_p,&p,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_eps,&eps,sizeof(double),hipMemcpyHostToDevice);
	
	//printf("step into kernel\n");
	queryKernel<<<1,size>>>(dev_nodeX,dev_nodeY,dev_result,dev_p,dev_eps); 
	//printf("step out of kernel\n");
	
	hipDeviceSynchronize();
	hipMemcpy(result,dev_result,size*sizeof(int),hipMemcpyDeviceToHost);
	
	//printf("%u:",total++);
	for(i=0;i<size;i++)
	{
		if(result[i]==-9999)
		{
			neighborhood.push(i);
			//printf("%d ",i);
		}
			
	}
	//printf("\n");
	
	free(result);free(nodeX);free(nodeY);
	hipFree(dev_nodeX);hipFree(nodeY);hipFree(dev_result);hipFree(dev_p);hipFree(dev_eps);
	
	return (int)neighborhood.size();
}
// expand cluster formed by p, which works in a way of bfs.
bool expand_cluster( vector<Point> &dataset, int p, int c, double eps, int min_pts){
    queue<int> neighbor_pts;
    dataset[p].lable = c;
	
    //printf("step into query1\n");
    region_query(dataset, p, neighbor_pts, eps);
	//printf("step out of query1\n");
    
    while (!neighbor_pts.empty()) {
        
        int neighbor = neighbor_pts.front();
        queue<int> neighbor_pts1;
		
		//printf("neighbor is %d\n",neighbor);
		//printf("step into query2\n");
        region_query(dataset, neighbor, neighbor_pts1, eps);
        //printf("step out of query2\n");
		
		
        if(neighbor_pts1.size()>=min_pts-1)
        {
            while (!neighbor_pts1.empty()) {
                
                int pt = neighbor_pts1.front();
                if(dataset[pt].lable ==-1){
                    neighbor_pts.push(pt);
                }
                neighbor_pts1.pop();
            }
        }
        dataset[neighbor].lable = c;
        neighbor_pts.pop();
        
    }
    return  true;
}

// doing dbscan, given radius and minimum number of neigborhoods.
int dbscan(vector<Point> &dataset, double eps, int min_pts)
{
    int c = 0;  // cluster lable
	int count = 0;
	int p;
    for (p = 0; p<dataset.size(); p++) {
        queue<int> neighborhood;
		printf("%d query start\n",count++);
        region_query(dataset, p, neighborhood, eps);//找到p的相邻节点，它们之间的距离小于eps，将相邻节点存放到队列中
        //printf("%d query end\n",count);
        if (neighborhood.size()+1 < min_pts) {//如果p相离节点与其组成团的大小 小于最小的要求，将其标记为0
            // mark as noise
			printf("miaomiaomiao?");
            dataset[p].lable = 0;
        }else
        {
           
            if(dataset[p].lable==-1){//否则，如果p没有被分类，则将其相邻节点扩展
                 c++;
				 //printf("step into cluster\n");
                expand_cluster(dataset, p,c, eps, min_pts);
				//printf("step out of cluster\n");
            }
        }
    }
    return c;
    
}