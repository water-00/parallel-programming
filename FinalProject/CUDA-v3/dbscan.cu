#include "hip/hip_runtime.h"
/*
1.���������ĺ�������
2.��1�Ż����޸���GPU���ظ�����ռ�Ĳ���
3.��region query���л�������������Ҫ����Ľ��ȫ�����м������֮��������鼴�ɵõ������

*/


#include "dbscan.h"
#include <math.h>
#include <queue>
#include<iostream>
#include "hip/hip_runtime.h"
#include ""
using namespace std;

/*
// calculate eculidean distance of two 2-D points
double euclidean_distance(Point a, Point b)
{
double x = a.x-b.x;
double y = a.y-b.y;
return sqrt(x*x+y*y);
}

// get neighborhood of point p and add it to neighborhood queue

int region_query( vector<Point> &dataset, int p, queue<int> &neighborhood, double eps)
{
	//int count = 0;
	for (int i = 0; i < dataset.size(); i++) {
		//cout << "regin_query" << count++ << endl;
		if(i!=p){
			int dist = euclidean_distance(dataset[p],dataset[i]);
			if ( dist< eps) {
				neighborhood.push(i);
			}
		}
	}
	return (int)neighborhood.size();
}
*/

unsigned int total = 0;
double* dev_nodeX;
double* dev_nodeY;
int* dev_result;
int* dev_p;
double* dev_eps;
int *result;
int datasize;
int** total_query_result;
//vector<int> *final_result;
//int queuesize;


/*
__device__ int cal(double *dev_nodeX, double *dev_nodeY, int* dev_p, int* dev_i,double* dev_eps)
{
	int i = *dev_i;
	if (i != *dev_p)
	{
		//int dist=euclidean_distance(dev_nodeX[i],dev_nodeY[i],dev_nodeX[*dev_p],dev_nodeY[*dev_p]);
		double x = dev_nodeX[i] - dev_nodeX[*dev_p];
		double y = dev_nodeY[i] - dev_nodeY[*dev_p];
		int dist = sqrt(x*x + y*y);
		//if (dist<*dev_eps) printf(" #%d (%.3f, %.3f) -> #%d(%.3f,%.3f) dist is %d\n",i, dev_nodeX[i], dev_nodeY[i], *dev_p, dev_nodeX[*dev_p], dev_nodeY[*dev_p], dist);
		if (dist<*dev_eps)
		{
			return 1;
		}
	}
	return 0;
}
*/
__global__ void region_query_kernal(double *dev_nodeX, double *dev_nodeY,int* dev_query_size, int* dev_query_target, int *dev_pointer, double* dev_eps)
{
	int size = *dev_query_size;
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	
	int index=bid*blockDim.x+tid;
	
	//printf("%d ah\n",index);
	
	
	int i=index/size;//��ǰ������ǵڼ��У��б��
	
	int j=index%size;//��ǰ������ǵڼ���Ԫ�أ��б��
		
	int target = dev_query_target[i];//��ǰ���������һ��Ŀ��Ԫ��
	
	//int value=cal(dev_nodeX,dev_nodeY,&target,&j,dev_eps);//���㵱ǰ�����Ԫ����Ŀ��Ԫ�صľ��룬�������Ҫ�󷵻�1�����򷵻�0
	
	int value=0;
	
	if (target != j)
	{
		//int dist=euclidean_distance(dev_nodeX[i],dev_nodeY[i],dev_nodeX[*dev_p],dev_nodeY[*dev_p]);
		double x = dev_nodeX[j] - dev_nodeX[target];
		double y = dev_nodeY[j] - dev_nodeY[target];
		int dist = sqrt(x*x + y*y);
		//if (dist<*dev_eps) printf(" #%d (%.3f, %.3f) -> #%d(%.3f,%.3f) dist is %d\n",i, dev_nodeX[i], dev_nodeY[i], *dev_p, dev_nodeX[*dev_p], dev_nodeY[*dev_p], dist);
		if (dist<*dev_eps)
		{
			value=1;
		}
	}
	
	if(value){//�����ǰ�����Ԫ�ط���Ҫ�����-9999�������1
		//printf("???\n");
		dev_pointer[index]=-9999;
	}
	else{
		dev_pointer[index]=1;
	}
}

void pral_query(int*query_target,int**total_query_result, double eps)
{
	
	//int *query_target;//�洢Ҫ�������Ԫ��
	//int **query_result;//���
	int *pointer;
	//hipError_t cudaStatus;
	
	int *dev_query_target;
	//int **dev_query_result;
	int *dev_pointer;
	int *dev_query_size;
	
	
	//hipMalloc((void***)&dev_query_result, datasize*sizeof(int*));
	hipMalloc((void**)&dev_pointer, datasize*datasize*sizeof(int));
	//query_result=(int**)malloc(datasize*sizeof(int*));
	pointer=(int*)malloc(datasize*datasize*sizeof(int));
	/*
	for(int i=0;i<datasize;i++)
	{
		query_result[i]=dev_pointer+i*datasize;
	}
	*/
	//hipMemcpy(dev_query_result, query_result, datasize*sizeof(int*), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&dev_query_target, datasize*sizeof(int));
	hipMalloc((void**)&dev_query_size, sizeof(int));
	
	
	hipMemcpy(dev_query_size, &datasize, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_query_target, query_target, datasize*sizeof(int), hipMemcpyHostToDevice);
	
	//printf("step into kernal\n");
	region_query_kernal<<<(datasize*datasize+511)/512,512>>>(dev_nodeX, dev_nodeY,dev_query_size, dev_query_target, dev_pointer, dev_eps);
	//printf("step out of kernal\n");
	//hipError_t error = hipGetLastError();
	//printf("CUDA error: %s\n", hipGetErrorString(error));
	
	hipDeviceSynchronize();

	hipMemcpy(pointer, dev_pointer, datasize*datasize*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int p=0;p<datasize*datasize;p++)
	{
		int i=p/datasize;
		int j=p%datasize;
		total_query_result[i][j]=pointer[p];
	}
	
	//free(query_result);
	free(pointer);
	hipFree(dev_query_target);
	//hipFree(dev_query_result);
	hipFree(dev_query_size);hipFree(dev_pointer);

		
}

// expand cluster formed by p, which works in a way of bfs.
bool expand_cluster(vector<Point> &dataset, int p, int c, double eps, int min_pts) {
	queue<int> neighbor_pts;
	dataset[p].lable = c;


	for(int i=0;i<datasize;i++)
	{
		if(total_query_result[p][i]==-9999)
			neighbor_pts.push(i);
	}

	
	while (!neighbor_pts.empty()) {

		int neighbor = neighbor_pts.front();
		queue<int> neighbor_pts1;

		//printf("neighbor is %d\n",neighbor);
		//printf("step into query2\n");
		//region_query(dataset, neighbor, neighbor_pts1, eps);
		//printf("step out of query2\n");
		for(int i=0;i<datasize;i++)
		{
			if(total_query_result[neighbor][i]==-9999)
				neighbor_pts1.push(i);
		}

		if (neighbor_pts1.size() >= min_pts - 1)
		{
			while (!neighbor_pts1.empty()) {

				int pt = neighbor_pts1.front();
				if (dataset[pt].lable == -1) {
					neighbor_pts.push(pt);
				}
				neighbor_pts1.pop();
			}
		}
		dataset[neighbor].lable = c;
		neighbor_pts.pop();

	}
	
	return  true;
}

void allocate_data_init(vector<Point> &dataset, double eps){
// device memory allocate
	int size = dataset.size();
	datasize=size;
	double *nodeX = (double*)malloc(size * sizeof(double));
	double *nodeY = (double*)malloc(size * sizeof(double));

	for (int i = 0; i<size; i++)
	{
		nodeX[i] = dataset[i].x;
		nodeY[i] = dataset[i].y;
	}
	
	hipMalloc((void**)&dev_nodeX, size * sizeof(double));
	hipMalloc((void**)&dev_nodeY, size * sizeof(double));
	hipMalloc((void**)&dev_eps, sizeof(double));
	hipMemcpy(dev_nodeX, nodeX, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_nodeY, nodeY, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_eps, &eps, sizeof(double), hipMemcpyHostToDevice);

	//hipMalloc((void**)&dev_result, size * sizeof(int));
	//hipMalloc((void**)&dev_p, sizeof(int));

	free(nodeX); free(nodeY);
//  host memory allocate
	//result = (int*)malloc(size * sizeof(int));
	
	//final_result=(vector<int>*)malloc(size*queuesize);
	
}

void allocate_data_free() {
// device memory free
	hipFree(dev_nodeX); hipFree(dev_nodeY); 
	//hipFree(dev_result);
	//hipFree(dev_p); 
	hipFree(dev_eps);
// host memory free
	//free(result);
	
	//free(final_result);
}

// doing dbscan, given radius and minimum number of neigborhoods.
int dbscan(vector<Point> &dataset, double eps, int min_pts)
{
	int c = 0;  // cluster lable
	//int count = 0;
	int p;
	
	int size=dataset.size();
	
	int* query_target=(int*) malloc(size*sizeof(int));
	
	for(int i=0;i<size;i++) query_target[i]=i;
	
	//queuesize=sizeof(query_target);
	
	allocate_data_init(dataset, eps);//�����Ѿ���dev_nodeX,dev_nodeY,dev_eps������GPU��
	
	
	//����ÿһ�ε���pral_query����������ѽ���������������Ȼ�������Ϊ��������
	//��������е�ÿһ�� ��Ӧtarget�е�һ��Ԫ�ص��ھ�
	total_query_result = (int**)malloc(size*sizeof(int*));
	for(int i=0;i<size;i++)
	{
		total_query_result[i]=(int*)malloc(size*sizeof(int));
	}
	//�������ÿ��Ԫ�ص��ھ�
	//printf("start to pral\n");
	clock_t start, finish;
	double duration;
	start = clock();
	
	pral_query(query_target,total_query_result, eps);//query_target�е�Ԫ�ر��߳�
	
	finish = clock();
	duration = (double)(finish - start) / CLOCKS_PER_SEC;
	cout << "pral: "<< duration << "s" << endl;
	//printf("finish to pral\n");
	free(query_target);
	
	
	for (p = 0; p<size; p++) {
		queue<int> neighborhood;
		//printf("%d query start\n", count++);
		for(int i=0;i<size;i++)
		{
			if(total_query_result[p][i]==-9999) neighborhood.push(i);
			//printf("%d done\n",i);
		}
		//printf("%d query end\n", count++);
		//region_query(dataset, p, neighborhood, eps);//�ҵ�p�����ڽڵ㣬����֮��ľ���С��eps�������ڽڵ��ŵ�������
													//printf("%d query end\n",count);
		if (neighborhood.size() + 1 < min_pts) {//���p����ڵ���������ŵĴ�С С����С��Ҫ�󣬽�����Ϊ0
												// mark as noise
			//printf("miaomiaomiao?");
			dataset[p].lable = 0;
		}
		else
		{

			if (dataset[p].lable == -1) {//�������pû�б����࣬�������ڽڵ���չ
				c++;
				//printf("step into cluster\n");
				expand_cluster(dataset, p, c, eps, min_pts);
				//printf("step out of cluster\n");
			}
		}
	}
	free(total_query_result);
	allocate_data_free();
	return c;
}