#include "hip/hip_runtime.h"
/*
1.���������ĺ�������
2.��1�Ż����޸���GPU���ظ�����ռ�Ĳ���
3.��region query���л�������������Ҫ����Ľ��ȫ�����м������֮��������鼴�ɵõ������

*/


#include "dbscan.h"
#include <math.h>
#include <queue>
#include<iostream>
#include "hip/hip_runtime.h"
#include ""
using namespace std;

/*
// calculate eculidean distance of two 2-D points
double euclidean_distance(Point a, Point b)
{
double x = a.x-b.x;
double y = a.y-b.y;
return sqrt(x*x+y*y);
}

// get neighborhood of point p and add it to neighborhood queue

int region_query( vector<Point> &dataset, int p, queue<int> &neighborhood, double eps)
{
	//int count = 0;
	for (int i = 0; i < dataset.size(); i++) {
		//cout << "regin_query" << count++ << endl;
		if(i!=p){
			int dist = euclidean_distance(dataset[p],dataset[i]);
			if ( dist< eps) {
				neighborhood.push(i);
			}
		}
	}
	return (int)neighborhood.size();
}
*/

unsigned int total = 0;
double* dev_nodeX;
double* dev_nodeY;
int* dev_result;
int* dev_p;
int *dev_label,*dev_elementsToAdd,*dev_elementsToOpe;
int *dev_query_size;
int **dev_total_query_result;
double* dev_eps;
int *dev_min_pts;
int *dev_pointer;
int *result;
int datasize;
int** total_query_result;
int *label,*elementsToAdd,*elementsToOpe;
//vector<int> *final_result;
//int queuesize;


/*
__device__ int cal(double *dev_nodeX, double *dev_nodeY, int* dev_p, int* dev_i,double* dev_eps)
{
	int i = *dev_i;
	if (i != *dev_p)
	{
		//int dist=euclidean_distance(dev_nodeX[i],dev_nodeY[i],dev_nodeX[*dev_p],dev_nodeY[*dev_p]);
		double x = dev_nodeX[i] - dev_nodeX[*dev_p];
		double y = dev_nodeY[i] - dev_nodeY[*dev_p];
		int dist = sqrt(x*x + y*y);
		//if (dist<*dev_eps) printf(" #%d (%.3f, %.3f) -> #%d(%.3f,%.3f) dist is %d\n",i, dev_nodeX[i], dev_nodeY[i], *dev_p, dev_nodeX[*dev_p], dev_nodeY[*dev_p], dist);
		if (dist<*dev_eps)
		{
			return 1;
		}
	}
	return 0;
}
*/
 __global__ void region_query_kernal(double *dev_nodeX, double *dev_nodeY,int* dev_query_size, int* dev_query_target, int *dev_pointer, double* dev_eps)
{
	//printf("???");
	int size = *dev_query_size;
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	
	int index=bid*blockDim.x+tid;
	
	//printf("%d ah\n",index);
	
	
	int i=index/size;//��ǰ������ǵڼ��У��б��
	
	int j=index%size;//��ǰ������ǵڼ���Ԫ�أ��б��
		
	int target = dev_query_target[i];//��ǰ���������һ��Ŀ��Ԫ��
	
	//int value=cal(dev_nodeX,dev_nodeY,&target,&j,dev_eps);//���㵱ǰ�����Ԫ����Ŀ��Ԫ�صľ��룬�������Ҫ�󷵻�1�����򷵻�0
	
	int value=0;
	
	if (target != j)
	{
		//int dist=euclidean_distance(dev_nodeX[i],dev_nodeY[i],dev_nodeX[*dev_p],dev_nodeY[*dev_p]);
		double x = dev_nodeX[j] - dev_nodeX[target];
		double y = dev_nodeY[j] - dev_nodeY[target];
		int dist = sqrt(x*x + y*y);
		//if (dist<*dev_eps) printf(" #%d (%.3f, %.3f) -> #%d(%.3f,%.3f) dist is %d\n",i, dev_nodeX[i], dev_nodeY[i], *dev_p, dev_nodeX[*dev_p], dev_nodeY[*dev_p], dist);
		if (dist<*dev_eps)
		{
			value=1;
		}
	}
	
	if(value){//�����ǰ�����Ԫ�ط���Ҫ�����-9999�������1
		//printf("???\n");
		dev_pointer[index]=-9999;
	}
	else{
		dev_pointer[index]=1;
	}
}

void pral_query(int*query_target,int**total_query_result, double eps)
{
	
	//int *query_target;//�洢Ҫ�������Ԫ��
	//int **query_result;//���
	int *pointer;
	//hipError_t cudaStatus;
	
	int *dev_query_target;
	//int **dev_query_result;
	
	
	
	
	//hipMalloc((void***)&dev_query_result, datasize*sizeof(int*));
	
	//query_result=(int**)malloc(datasize*sizeof(int*));
	pointer=(int*)malloc(datasize*datasize*sizeof(int));
	/*
	for(int i=0;i<datasize;i++)
	{
		query_result[i]=dev_pointer+i*datasize;
	}
	*/
	//hipMemcpy(dev_query_result, query_result, datasize*sizeof(int*), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&dev_query_target, datasize*sizeof(int));
	
	
	
	
	hipMemcpy(dev_query_target, query_target, datasize*sizeof(int), hipMemcpyHostToDevice);
	
	//printf("step into kernal\n");
	region_query_kernal<<<(datasize*datasize+511)/512,512>>>(dev_nodeX, dev_nodeY,dev_query_size, dev_query_target, dev_pointer, dev_eps);
	//printf("step out of kernal\n");
	//hipError_t error = hipGetLastError();
	//printf("CUDA error: %s\n", hipGetErrorString(error));
	
	hipDeviceSynchronize();

	hipMemcpy(pointer, dev_pointer, datasize*datasize*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int p=0;p<datasize*datasize;p++)
	{
		int i=p/datasize;
		int j=p%datasize;
		total_query_result[i][j]=pointer[p];
	}
	
	//free(query_result);
	free(pointer);
	hipFree(dev_query_target);
	//hipFree(dev_query_result);
	

		
}


__global__ void expand_cluster_kernal(int *dev_query_size,int *dev_pointer,int* dev_elementsToAdd,int *dev_elementsToOpe,int *dev_label,int *dev_min_pts)
{
	int index=threadIdx.x;
	int size=*dev_query_size;
	int target=dev_elementsToOpe[index];
	int counter=0;
	for(int i=0;i<size;i++)
	{
		int location = target*size+i;
		if(dev_pointer[location]==-9999)
			counter++;
	}
	if(counter>=*dev_min_pts-1)
	{
		for(int i=0;i<size;i++)
		{
			int location = target*size+i;
			if(dev_pointer[location]==-9999&&dev_label[i]==-1)
			{
				dev_elementsToAdd[i]=-9999;
			}
		}
	}
}

bool expand_cluster(vector<Point> &dataset, int p, int c, double eps, int min_pts) {
	queue<int> neighbor_pts;
	//queue<int> neighbor_ope;//�ṩ��pral_query���������߳�
	dataset[p].lable = c;

	for(int i=0;i<datasize;i++)
	{
		if(total_query_result[p][i]==-9999)
			neighbor_pts.push(i);
	}

	
	int countOfOpe=0;
	while (!neighbor_pts.empty()) {
			countOfOpe=0;
			while(!neighbor_pts.empty())
			{
				int t=neighbor_pts.front();
				elementsToOpe[countOfOpe++]=t;
				neighbor_pts.pop();
				dataset[t].lable=c;
			}
			
			for(int i=0;i<datasize;i++)
			{
				label[i]=dataset[i].lable;
			}
			
			hipMemcpy(dev_elementsToAdd, elementsToAdd, datasize*sizeof(int), hipMemcpyHostToDevice);//��ʼ��������飬����ϴβ�����Ӱ�죻
			hipMemcpy(dev_elementsToOpe, elementsToOpe, datasize*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(dev_label, label, datasize * sizeof(int), hipMemcpyHostToDevice);
			
			expand_cluster_kernal<<<1,countOfOpe>>>(dev_query_size,dev_pointer,dev_elementsToAdd,dev_elementsToOpe,dev_label,dev_min_pts);
		
			hipMemcpy(elementsToAdd, dev_elementsToAdd, datasize*sizeof(int), hipMemcpyDeviceToHost);
			
			for(int i=0;i<datasize;i++)
			{
				if(elementsToAdd[i]==-9999)
				{
					neighbor_pts.push(i);
					elementsToAdd[i]=1;
				}
					
			}
			
	}
	
	return  true;
}

void allocate_data_init(vector<Point> &dataset, double eps){
// device memory allocate
	int size = dataset.size();
	datasize=size;
	double *nodeX = (double*)malloc(size * sizeof(double));
	double *nodeY = (double*)malloc(size * sizeof(double));

	for (int i = 0; i<size; i++)
	{
		nodeX[i] = dataset[i].x;
		nodeY[i] = dataset[i].y;
	}
	
	hipMalloc((void**)&dev_nodeX, size * sizeof(double));
	hipMalloc((void**)&dev_nodeY, size * sizeof(double));
	hipMalloc((void**)&dev_eps, sizeof(double));
	hipMemcpy(dev_nodeX, nodeX, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_nodeY, nodeY, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_eps, &eps, sizeof(double), hipMemcpyHostToDevice);

	//hipMalloc((void**)&dev_result, size * sizeof(int));
	//hipMalloc((void**)&dev_p, sizeof(int));
	hipMalloc((void**)&dev_pointer, datasize*datasize*sizeof(int));
	

	free(nodeX); free(nodeY);
	
	hipMalloc((void**)&dev_query_size, sizeof(int));
	hipMemcpy(dev_query_size, &datasize, sizeof(int), hipMemcpyHostToDevice);
	
//  host memory allocate
	//result = (int*)malloc(size * sizeof(int));
	
	//final_result=(vector<int>*)malloc(size*queuesize);
	
}

void allocate_data_free() {
// device memory free
	hipFree(dev_nodeX); hipFree(dev_nodeY); 
	//hipFree(dev_result);
	//hipFree(dev_p); 
	hipFree(dev_eps);
	
	hipFree(dev_pointer);
	
	hipFree(dev_query_size);
// host memory free
	//free(result);
	
	//free(final_result);
}




void allocate_data_init2(vector<Point> &dataset, double eps,int min_pts){
// device memory allocate

	label = (int*)malloc(datasize * sizeof(int));//label���ÿ��Ԫ�ض�Ӧ��lable

	for (int i = 0; i<datasize; i++)
	{
		label[i] = dataset[i].lable;
	}
	
	

	hipMalloc((void**)&dev_label, datasize * sizeof(int));
	hipMemcpy(dev_label, label, datasize * sizeof(int), hipMemcpyHostToDevice);
	
	hipMalloc((void**)&dev_min_pts, sizeof(int));
	hipMemcpy(dev_min_pts, &min_pts, sizeof(int), hipMemcpyHostToDevice);

	elementsToAdd = (int*)malloc(datasize*sizeof(int));//elementsToAdd����´η��������Ҫ������չ�ĵ�
	for(int i=0;i<datasize;i++) elementsToAdd[i]=1;
	hipMalloc((void**)&dev_elementsToAdd, datasize * sizeof(int));
	
	elementsToOpe = (int*)malloc(datasize*sizeof(int));//elementsToOpe���Ҫ������չ�ĵ�
	hipMalloc((void**)&dev_elementsToOpe, datasize * sizeof(int));
	
	//hipMalloc((void**)&dev_result, size * sizeof(int));
	//hipMalloc((void**)&dev_p, sizeof(int));

//  host memory allocate
	//result = (int*)malloc(size * sizeof(int));
	
	//final_result=(vector<int>*)malloc(size*queuesize);
	
}

void allocate_data_free2() {
// device memory free
	hipFree(dev_label); hipFree(dev_elementsToAdd); hipFree(dev_elementsToOpe);hipFree(dev_min_pts);
	//hipFree(dev_result);
	//hipFree(dev_p); 
// host memory free
	//free(result);
	free(label);free(elementsToAdd);free(elementsToOpe);
	//free(final_result);
}





// doing dbscan, given radius and minimum number of neigborhoods.
int dbscan(vector<Point> &dataset, double eps, int min_pts)
{
	int c = 0;  // cluster lable
	//int count = 0;
	int p;
	
	int size=dataset.size();
	
	int* query_target=(int*) malloc(size*sizeof(int));
	
	for(int i=0;i<size;i++) query_target[i]=i;
	
	//queuesize=sizeof(query_target);
	
	allocate_data_init(dataset, eps);//�����Ѿ���dev_nodeX,dev_nodeY,dev_eps������GPU��
	allocate_data_init2(dataset, eps, min_pts);
	
	//����ÿһ�ε���pral_query����������ѽ���������������Ȼ�������Ϊ��������
	//��������е�ÿһ�� ��Ӧtarget�е�һ��Ԫ�ص��ھ�
	total_query_result = (int**)malloc(size*sizeof(int*));
	for(int i=0;i<size;i++)
	{
		total_query_result[i]=(int*)malloc(size*sizeof(int));
	}
	//�������ÿ��Ԫ�ص��ھ�
	//printf("start to pral\n");
	//clock_t start, finish;
	//double duration;
	//start = clock();
	
	pral_query(query_target,total_query_result, eps);//query_target�е�Ԫ�ر��߳�
	
	//finish = clock();
	//duration = (double)(finish - start) / CLOCKS_PER_SEC;
	//cout << "pral: "<< duration << "s" << endl;
	//printf("finish to pral\n");
	free(query_target);
	
	
	for (p = 0; p<size; p++) {
		queue<int> neighborhood;
		//printf("%d query start\n", count++);
		for(int i=0;i<size;i++)
		{
			if(total_query_result[p][i]==-9999) neighborhood.push(i);
			//printf("%d done\n",i);
		}
		//printf("%d query end\n", count++);
		//region_query(dataset, p, neighborhood, eps);//�ҵ�p�����ڽڵ㣬����֮��ľ���С��eps�������ڽڵ��ŵ�������
													//printf("%d query end\n",count);
		if (neighborhood.size() + 1 < min_pts) {//���p����ڵ���������ŵĴ�С С����С��Ҫ�󣬽�����Ϊ0
												// mark as noise
			//printf("miaomiaomiao?");
			dataset[p].lable = 0;
		}
		else
		{

			if (dataset[p].lable == -1) {//�������pû�б����࣬�������ڽڵ���չ
				c++;
				printf("step into cluster %d\n",p);
				expand_cluster(dataset, p, c, eps, min_pts);
				printf("step out of cluster %d\n",p);
			}
		}
	}
	free(total_query_result);
	allocate_data_free();
	allocate_data_free2();
	return c;
}